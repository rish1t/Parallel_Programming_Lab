#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void matmul_rowwise(int *a, int *b, int *t, int m, int n, int p)
{
    int row = threadIdx.x;
    if (row < m)
    {
        for (int col = 0; col < p; col++)
        {
            t[row * p + col] = 0;
            for (int k = 0; k < n; k++)
            {
                t[row * p + col] += a[row * n + k] * b[k * p + col];
            }
        }
    }
}

__global__ void matmul_columnwise(int *a, int *b, int *t, int m, int n, int p)
{
    int col = threadIdx.x;
    if (col < p)
    {
        for (int row = 0; row < m; row++)
        {
            t[row * p + col] = 0;
            for (int k = 0; k < n; k++)
            {
                t[row * p + col] += a[row * n + k] * b[k * p + col];
            }
        }
    }
}

__global__ void matmul_elementwise(int *a, int *b, int *t, int m, int n, int p)
{
    int row = threadIdx.x;
    int col = threadIdx.y;
    if (row < m && col < p)
    {
        t[row * p + col] = 0;
        for (int k = 0; k < n; k++)
        {
            t[row * p + col] += a[row * n + k] * b[k * p + col];
        }
    }
}

int check_multiplication_dimensions(int m1, int n1, int m2, int n2)
{
    if (n1 != m2)
    {
        printf("Error: Matrices dimensions must match for multiplication (columns of A == rows of B).\n");
        return 0;
    }
    return 1;
}

int main(void)
{
    int *a, *b, *c;
    int m, n, p;
    int *d_a, *d_b, *d_c;

    printf("Enter the dimensions of matrix A (rows x columns): ");
    scanf("%d %d", &m, &n);

    a = (int *)malloc(m * n * sizeof(int));
    printf("Enter matrix A:\n");
    for (int i = 0; i < m * n; i++)
        scanf("%d", &a[i]);

    printf("Enter the dimensions of matrix B (rows x columns): ");
    int m2, n2;
    scanf("%d %d", &m2, &n2);

    if (!check_multiplication_dimensions(m, n, m2, n2))
    {
        free(a);
        return -1;
    }

    b = (int *)malloc(m2 * n2 * sizeof(int));
    printf("Enter matrix B:\n");
    for (int i = 0; i < m2 * n2; i++)
        scanf("%d", &b[i]);

    int size_a = sizeof(int) * m * n;
    int size_b = sizeof(int) * m2 * n2;
    int size_c = sizeof(int) * m * n2;

    hipMalloc((void **)&d_a, size_a);
    hipMalloc((void **)&d_b, size_b);
    hipMalloc((void **)&d_c, size_c);

    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

    matmul_rowwise<<<1, m>>>(d_a, d_b, d_c, m, n, n2);

    matmul_columnwise<<<1, n2>>>(d_a, d_b, d_c, m, n, n2);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((m + 15) / 16, (n2 + 15) / 16);
    matmul_elementwise<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, m, n, n2);

    c = (int *)malloc(m * n2 * sizeof(int));
    hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);

    printf("\nMatrix multiplication result (row-wise):\n");
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n2; j++)
            printf("%d\t", c[i * n2 + j]);
        printf("\n");
    }

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
