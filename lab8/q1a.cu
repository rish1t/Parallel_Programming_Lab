#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void matadd_rowwise(int *a, int *b, int *t, int m, int n)
{
    int row = threadIdx.x;
    if (row < m)
    {
        for (int col = 0; col < n; col++)
        {
            t[row * n + col] = a[row * n + col] + b[row * n + col];
        }
    }
}

__global__ void matadd_columnwise(int *a, int *b, int *t, int m, int n)
{
    int col = threadIdx.x;
    if (col < n)
    {
        for (int row = 0; row < m; row++)
        {
            t[row * n + col] = a[row * n + col] + b[row * n + col];
        }
    }
}

__global__ void matadd_elementwise(int *a, int *b, int *t, int m, int n)
{
    int row = threadIdx.x;
    int col = threadIdx.y;
    if (row < m && col < n)
    {
        t[row * n + col] = a[row * n + col] + b[row * n + col];
    }
}

int check_addition_dimensions(int m1, int n1, int m2, int n2)
{
    if (m1 != m2 || n1 != n2)
    {
        printf("Error: Matrices dimensions must match for addition.\n");
        return 0;
    }
    return 1;
}

int main(void)
{
    int *a, *b, *t;
    int m, n;
    int *d_a, *d_b, *d_t;

    printf("Enter the dimensions of matrix A (rows x columns): ");
    scanf("%d %d", &m, &n);

    a = (int *)malloc(m * n * sizeof(int));
    printf("Enter matrix A:\n");
    for (int i = 0; i < m * n; i++)
        scanf("%d", &a[i]);

    printf("Enter the dimensions of matrix B (rows x columns): ");
    int m2, n2;
    scanf("%d %d", &m2, &n2);

    if (!check_addition_dimensions(m, n, m2, n2))
    {
        free(a);
        return -1;
    }

    b = (int *)malloc(m2 * n2 * sizeof(int));
    printf("Enter matrix B:\n");
    for (int i = 0; i < m2 * n2; i++)
        scanf("%d", &b[i]);

    int size_a = sizeof(int) * m * n;
    int size_b = sizeof(int) * m2 * n2;
    int size_t = sizeof(int) * m * n;

    hipMalloc((void **)&d_a, size_a);
    hipMalloc((void **)&d_b, size_b);
    hipMalloc((void **)&d_t, size_t); // For result

    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

    matadd_rowwise<<<1, m>>>(d_a, d_b, d_t, m, n);

    //matadd_columnwise<<<1, n>>>(d_a, d_b, d_t, m, n);

    //dim3 threadsPerBlock(16, 16);
    //dim3 numBlocks((m + 15) / 16, (n + 15) / 16);
    //matadd_elementwise<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_t, m, n);

    t = (int *)malloc(m * n * sizeof(int));
    hipMemcpy(t, d_t, size_t, hipMemcpyDeviceToHost);

    printf("\nMatrix addition result (row-wise):\n");
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
            printf("%d\t", t[i * n + j]);
        printf("\n");
    }

    free(a);
    free(b);
    free(t);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_t);

    return 0;
}
