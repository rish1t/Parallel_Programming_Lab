#include "hip/hip_runtime.h"

#include <stdio.h>
// #include <math_functions.h>

__device__ int getGTID()
{
    int gtid = blockIdx.x * blockDim.x + threadIdx.x + 
               blockIdx.y * blockDim.x * gridDim.x + 
               threadIdx.y * blockDim.x;
    return gtid;
}

__global__ void compute_sine(float *da, float *dc, int n)
{
    int gtid = getGTID();
    if (gtid < n)
    {
        dc[gtid] = sin(da[gtid]);
    }
}

int main()
{
    int n = 360 / 5;
    int t = 256;
    
    float *a = (float*)malloc(n * sizeof(float));
    float *c = (float*)malloc(n * sizeof(float));
    float *da, *dc;

    hipMalloc((void **)&da, n * sizeof(float));
    hipMalloc((void **)&dc, n * sizeof(float));

    for (int i = 0; i < n; i++) {
        a[i] = (float)(i * 5) * (M_PI / 180); 
    }

    hipMemcpy(da, a, n * sizeof(float), hipMemcpyHostToDevice);

    compute_sine<<<(n + t - 1) / t, t>>>(da, dc, n);

    hipMemcpy(c, dc, n * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
        printf("sin(%d degrees) = %f\n", i * 5, c[i]);

    printf("Unused threads: %d\n220905390\n", n > t ? t - (n % t) : t - n);

    free(a);
    free(c);
    hipFree(da);
    hipFree(dc);

    return 0;
}
