#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void add_vec(int*da, int*db, int*dc){
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    dc[index] = da[index] + db[index];
}

int main(){
    int n;
    printf("Length of the vector : ");
    scanf("%d",&n);

    int a[n],b[n],c[n];
    int *da,*db,*dc;

    hipMalloc((void **)&da,n*sizeof(int));
    hipMalloc((void **)&db,n*sizeof(int));
    hipMalloc((void **)&dc,n*sizeof(int));

    printf("Enter vector one : ");
    for(int i=0;i<n;i++)
        scanf("%d",&a[i]);

    printf("Enter vector two : ");
    for(int i=0;i<n;i++)
        scanf("%d",&b[i]);
    
    hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(db,b,n*sizeof(int),hipMemcpyHostToDevice);


    add_vec<<<n,1>>>(da,db,dc);
    hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<n;i++)
        printf("%d\t",c[i]);
    printf("\n220905390\n");
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}