#include "hip/hip_runtime.h"

#include <stdio.h>

__device__ int getGTID()
{
    int gtid = blockIdx.x * blockDim.x + threadIdx.x + 
               blockIdx.y * blockDim.x * gridDim.x + 
               threadIdx.y * blockDim.x;
    return gtid;
}

__global__ void add_vec(int *da, int *db, int *dc, int n)
{
    int gtid = getGTID();
    if (gtid < n)
    {
        dc[gtid] = da[gtid] + db[gtid];
    }
}

int main()
{
    int n, t = 256;
    printf("Length of the vector: ");
    scanf("%d", &n);

    int *a = (int*)malloc(n * sizeof(int));
    int *b = (int*)malloc(n * sizeof(int));
    int *c = (int*)malloc(n * sizeof(int));
    int *da, *db, *dc;

    hipMalloc((void **)&da, n * sizeof(int));
    hipMalloc((void **)&db, n * sizeof(int));
    hipMalloc((void **)&dc, n * sizeof(int));

    for (int i = 0; i < n; i++) {
        a[i] = i + 1;
        b[i] = i + 1;
    }

    hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n * sizeof(int), hipMemcpyHostToDevice);

    add_vec<<<ceil((float)n/256), t>>>(da, db, dc, n);

    hipMemcpy(c, dc, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
        printf("%d\t", c[i]);
    printf("\n");

    printf("Unused threads: %d\n220905390\n", n > t ? t-(n%t) : t - n);

    free(a);
    free(b);
    free(c);
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;
}
