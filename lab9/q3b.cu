#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__device__ void getOnesCompInBin(int* n, int* res) {
    int numBits = 0;
    int d = *n;
    while (d > 0) {
        numBits++;
        d >>= 1;
    }
    int oc = 0;
    for (int i = numBits - 1; i >= 0; i--) {
        int bit = ((*n) >> i) & 1;
        int comp = bit ? 0 : 1;
        oc = oc * 10 + comp;
    }
    *res = oc;
}

__global__ void replaceElements(int* d_A, int* d_B, int m, int n) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (r >= m || c >= n) return; 

    if (r > 0 && c > 0 && r < m - 1 && c < n - 1) {
        getOnesCompInBin(&d_A[r * n + c], &d_B[r * n + c]);
    } else {
        d_B[r * n + c] = d_A[r * n + c]; 
    }
}

int main() {
    int m, n;
    printf("Enter dimensions\n");
    scanf("%d %d", &m, &n);
    int A[m][n];
    int B[m][n];
    printf("Enter matrix\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            scanf("%d", &A[i][j]);
        }
    }
    int* d_A, *d_B;
    hipMalloc((void**) &d_A, m * n * sizeof(int));
    hipMalloc((void**) &d_B, m * n * sizeof(int));
    hipMemcpy(d_A, A, m * n * sizeof(int), hipMemcpyHostToDevice);
    dim3 dimGrid(ceil(n / 16.0), ceil(m / 16.0), 1);
    dim3 dimBlock(16, 16, 1);
    replaceElements<<<dimGrid, dimBlock>>>(d_A, d_B, m, n);
    hipMemcpy(B, d_B, m * n * sizeof(int), hipMemcpyDeviceToHost);
    printf("Result\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
}
