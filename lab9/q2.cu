#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void transformMatrix(int *A, int *B, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        int val = A[row * n + col];
        int power = row + 1;
        int result = 1;
        
        for (int i = 0; i < power; i++) {
            result *= val;
        }
        
        B[row * n + col] = result;
    }
}

void printMatrix(int *M, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d\t", M[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {
    int m, n;
    
    printf("Enter number of rows (M): ");
    scanf("%d", &m);
    
    printf("Enter number of columns (N): ");
    scanf("%d", &n);
    
    int *A = (int*)malloc(m * n * sizeof(int));
    int *B = (int*)malloc(m * n * sizeof(int));
    
    printf("Enter the elements of matrix A (%dx%d):\n", m, n);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            scanf("%d", &A[i * n + j]);
        }
    }
    
    printf("Matrix :\n");
    printMatrix(A, m, n);
    
    int size = m * n * sizeof(int);
    int *d_A, *d_B;
    
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    
    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
    
    transformMatrix<<<gridSize, blockSize>>>(d_A, d_B, m, n);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
    
    printf("Matrix after Operation:\n");
    printMatrix(B, m, n);
    
    hipFree(d_A);
    hipFree(d_B);
    free(A);
    free(B);
    
    return 0;
}