#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void spmv_csr_kernel(int num_rows, const int *row_ptr, const int *col_idx, 
                              const float *values, const float *x, float *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < num_rows) {
        float dot = 0.0f;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row + 1];
        
        for (int i = row_start; i < row_end; i++) {
            dot += values[i] * x[col_idx[i]];
        }
        
        y[row] = dot;
    }
}

int main() {
    int num_rows, num_cols, nnz;
    
    printf("Enter number of rows: ");
    scanf("%d", &num_rows);
    
    printf("Enter number of columns: ");
    scanf("%d", &num_cols);
    
    printf("Enter number of non-zero elements: ");
    scanf("%d", &nnz);
    
    int *h_row_ptr = (int*)malloc((num_rows + 1) * sizeof(int));
    int *h_col_idx = (int*)malloc(nnz * sizeof(int));
    float *h_values = (float*)malloc(nnz * sizeof(float));
    float *h_x = (float*)malloc(num_cols * sizeof(float));
    float *h_y = (float*)malloc(num_rows * sizeof(float));
    
    printf("Enter row_ptr array (%d values): ", num_rows + 1);
    for (int i = 0; i <= num_rows; i++) {
        scanf("%d", &h_row_ptr[i]);
    }
    
    printf("Enter col_idx array (%d values): ", nnz);
    for (int i = 0; i < nnz; i++) {
        scanf("%d", &h_col_idx[i]);
    }
    
    printf("Enter values array (%d values): ", nnz);
    for (int i = 0; i < nnz; i++) {
        scanf("%f", &h_values[i]);
    }
    
    printf("Enter input vector x (%d values): ", num_cols);
    for (int i = 0; i < num_cols; i++) {
        scanf("%f", &h_x[i]);
    }
    
    for (int i = 0; i < num_rows; i++) {
        h_y[i] = 0.0f;
    }
    
    int *d_row_ptr, *d_col_idx;
    float *d_values, *d_x, *d_y;
    
    hipMalloc((void**)&d_row_ptr, (num_rows + 1) * sizeof(int));
    hipMalloc((void**)&d_col_idx, nnz * sizeof(int));
    hipMalloc((void**)&d_values, nnz * sizeof(float));
    hipMalloc((void**)&d_x, num_cols * sizeof(float));
    hipMalloc((void**)&d_y, num_rows * sizeof(float));
    
    hipMemcpy(d_row_ptr, h_row_ptr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, h_col_idx, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, h_values, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, num_cols * sizeof(float), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_rows + threadsPerBlock - 1) / threadsPerBlock;
    
    spmv_csr_kernel<<<blocksPerGrid, threadsPerBlock>>>(num_rows, d_row_ptr, d_col_idx, d_values, d_x, d_y);
    
    hipMemcpy(h_y, d_y, num_rows * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("\nOutput vector y = A*x: ");
    for (int i = 0; i < num_rows; i++) {
        printf("%.1f ", h_y[i]);
    }
    printf("\n");
    
    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);
    
    free(h_row_ptr);
    free(h_col_idx);
    free(h_values);
    free(h_x);
    free(h_y);
    
    return 0;
}