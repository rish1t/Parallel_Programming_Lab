#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void conv_1D(float *N, float *M, float *P, int M_width, int width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= width)
        return;

    float Pval = 0.0;
    int start = i - (M_width / 2);
    for (int j = 0; j < M_width; j++)
    {
        int idx = start + j;
        if (idx >= 0 && idx < width)
        {
            Pval += N[idx] * M[j];
        }
    }
    P[i] = Pval;
}

int main()
{
    int n, m;
    printf("Enter the length of the Vector: ");
    scanf("%d", &n);
    printf("Enter length of the mask: ");
    scanf("%d", &m);

    float *N = (float *)malloc(n * sizeof(float));
    float *M = (float *)malloc(m * sizeof(float));
    float *P = (float *)malloc(n * sizeof(float));

    float *da, *db, *dc;

    hipMalloc((void **)&da, n * sizeof(float));
    hipMalloc((void **)&db, m * sizeof(float));
    hipMalloc((void **)&dc, n * sizeof(float));

    printf("Enter vector: ");
    for (int i = 0; i < n; i++)
        scanf("%f", &N[i]);

    printf("Enter Mask: ");
    for (int i = 0; i < m; i++)
        scanf("%f", &M[i]);

    hipMemcpy(da, N, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, M, m * sizeof(float), hipMemcpyHostToDevice);

    conv_1D<<<1, 256>>>(da, db, dc, m, n);

    hipMemcpy(P, dc, n * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
        printf("%2.2f\t", P[i]);
    printf("\n");

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    free(N);
    free(M);
    free(P);

    return 0;
}
