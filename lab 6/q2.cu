#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void selecsort(int *arr, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid >= n) return;

    for (int i = 0; i < n - 1; i++)
    {
        int minIdx = i;
        for (int j = i + 1; j < n; j++)
        {
            if (arr[j] < arr[minIdx])
            {
                minIdx = j;
            }
        }

        if (tid == i)
        {
            int temp = arr[i];
            arr[i] = arr[minIdx];
            arr[minIdx] = temp;
        }
        __syncthreads();
    }
}

int main()
{
    int n, i, *d_arr;
    printf("Enter the no. of elements : ");
    scanf("%d", &n);

    int arr[n], res[n];
    printf("Enter the array : ");
    for (i = 0; i < n; i++)
    {
        scanf("%d", &arr[i]);
    }

    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    selecsort<<<1, n>>>(d_arr, n);

    hipMemcpy(res, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array: \n");
    for (int i = 0; i < n; i++)
    {
        printf("%d ", res[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}
