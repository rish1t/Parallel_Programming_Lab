#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void oddEvenSort(int *arr, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int temp;

    if (tid >= n) return;

    for (int phase = 0; phase < n; phase++) {
        if (phase % 2 == 0) {
            if (tid % 2 == 0 && tid + 1 < n) {
                if (arr[tid] > arr[tid + 1]) {
                    temp = arr[tid];
                    arr[tid] = arr[tid + 1];
                    arr[tid + 1] = temp;
                }
            }
        } else {
            if (tid % 2 == 1 && tid + 1 < n) {
                if (arr[tid] > arr[tid + 1]) {
                    temp = arr[tid];
                    arr[tid] = arr[tid + 1];
                    arr[tid + 1] = temp;
                }
            }
        }
        __syncthreads();
    }
}



int main()
{
    int n, i, *d_arr;
    printf("Enter the no. of elements : ");
    scanf("%d", &n);

    int arr[n], res[n];
    printf("Enter the array : ");
    for (i = 0; i < n; i++)
    {
        scanf("%d", &arr[i]);
    }

    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    oddEvenSort<<<1, n>>>(d_arr, n);

    hipMemcpy(res, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array: \n");
    for (int i = 0; i < n; i++)
    {
        printf("%d ", res[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}
