#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void matmul(int *a, int *b, int *t, int m, int n, int q) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < m && c < q) {  // Ensure valid thread range
        int sum = 0;
        for (int k = 0; k < n; k++)
            sum += a[r * n + k] * b[k * q + c];
        t[r * q + c] = sum;
    }
}

int main(void) {
    int *a, *b, *t, m, n, p, q;
    int *d_a, *d_b, *d_t;

    printf("m value: ");
    scanf("%d", &m);
    printf("n value: ");
    scanf("%d", &n);
    printf("p value: ");
    scanf("%d", &p);
    printf("q value: ");
    scanf("%d", &q);

    if (n != p) {
        printf("Matrix multiplication not possible: n != p.\n");
        return -1;
    }

    int sizeA = sizeof(int) * m * n;
    int sizeB = sizeof(int) * p * q;
    int sizeT = sizeof(int) * m * q;

    a = (int *)malloc(sizeA);
    b = (int *)malloc(sizeB);
    t = (int *)malloc(sizeT);

    printf("Enter matrix A: ");
    for (int i = 0; i < m * n; i++)
        scanf("%d", &a[i]);

    printf("Enter matrix B: ");
    for (int i = 0; i < p * q; i++)
        scanf("%d", &b[i]);

    hipMalloc((void **)&d_a, sizeA);
    hipMalloc((void **)&d_b, sizeB);
    hipMalloc((void **)&d_t, sizeT);

    hipMemcpy(d_a, a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeB, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid(ceil((float)q / block.x), ceil((float)m / block.y));

    matmul<<<grid, block>>>(d_a, d_b, d_t, m, n, q);
    hipMemcpy(t, d_t, sizeT, hipMemcpyDeviceToHost);

    printf("Result matrix:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < q; j++)
            printf("%d ", t[i * q + j]);
        printf("\n");
    }

    free(a);
    free(b);
    free(t);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_t);

    return 0;
}
