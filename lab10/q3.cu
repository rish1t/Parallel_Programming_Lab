#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void convolution_1D_tiled(float *N, float *M, float *P, int width, int mask_width) {
    __shared__ float N_s[TILE_SIZE + 4];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int n = mask_width / 2;
    int halo_index = i - n;
    if (halo_index >= 0 && halo_index < width) {
        N_s[threadIdx.x] = N[halo_index];
    } else {
        N_s[threadIdx.x] = 0.0f;
    }
    __syncthreads();
    float Pvalue = 0.0f;
    if (threadIdx.x < TILE_SIZE && i < width) {
        for (int j = 0; j < mask_width; j++) {
            int index = threadIdx.x + j;
            if (index < TILE_SIZE + mask_width - 1) {
                Pvalue += N_s[index] * M[j];
            }
        }
        P[i] = Pvalue;
    }
}

int main() {
    int width, mask_width;
    printf("Enter the size of the input array: ");
    scanf("%d", &width);
    printf("Enter the size of the mask array: ");
    scanf("%d", &mask_width);
    float *h_N = (float*)malloc(width * sizeof(float));
    float *h_M = (float*)malloc(mask_width * sizeof(float));
    float *h_P = (float*)malloc(width * sizeof(float));
    printf("Enter input array elements: ");
    for (int i = 0; i < width; i++) scanf("%f", &h_N[i]);
    printf("Enter mask array elements: ");
    for (int i = 0; i < mask_width; i++) scanf("%f", &h_M[i]);
    float *d_N, *d_M, *d_P;
    hipMalloc(&d_N, width * sizeof(float));
    hipMalloc(&d_M, mask_width * sizeof(float));
    hipMalloc(&d_P, width * sizeof(float));
    hipMemcpy(d_N, h_N, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, mask_width * sizeof(float), hipMemcpyHostToDevice);
    int block_size = TILE_SIZE;
    int grid_size = (width + block_size - 1) / block_size;
    convolution_1D_tiled<<<grid_size, block_size>>>(d_N, d_M, d_P, width, mask_width);
    hipMemcpy(h_P, d_P, width * sizeof(float), hipMemcpyDeviceToHost);
    printf("Resultant array: ");
    for (int i = 0; i < width; i++) printf("%f ", h_P[i]);
    printf("\n");
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
    free(h_N);
    free(h_M);
    free(h_P);
    return 0;
}
