#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 5
#define K 5     

__constant__ float d_kernel[K];

__global__ void convolution1D(float *d_input, float *d_output, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float sum = 0.0f;
    
    if (i < size) {
        for (int j = 0; j < K; j++) {
            int index = i + j - K / 2;
            if (index >= 0 && index < size) {
                sum += d_input[index] * d_kernel[j];
            }
        }
        d_output[i] = sum;
    }
}

int main() {
    float h_input[N], h_output[N], h_kernel[K];
    float *d_input, *d_output;
    
    printf("Enter %d elements for input array:\n", N);
    for (int i = 0; i < N; i++) scanf("%f", &h_input[i]);

    printf("Enter %d elements for kernel:\n", K);
    for (int i = 0; i < K; i++) scanf("%f", &h_kernel[i]);

    hipMalloc((void**)&d_input, N * sizeof(float));
    hipMalloc((void**)&d_output, N * sizeof(float));

    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, K * sizeof(float));

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    convolution1D<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);

    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    
    printf("Convolution Output:\n");
    for (int i = 0; i < N; i++) {
        printf("h_output[%d] = %f\n", i, h_output[i]);
    }
    
    return 0;
}
