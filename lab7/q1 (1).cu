#include <iostream>
#include <string>
#include <hip/hip_runtime.h>

__global__ void countOccurrences(const char* s, const char* w, int* c, int slen, int wlen) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < slen - wlen + 1) {
        bool match = true;
        for (int i = 0; i < wlen; ++i) {
            if (s[idx + i] != w[i]) {
                match = false;
                break;
            }
        }
        if (match) atomicAdd(c, 1);
    }
}

int main() {
    char s[50], w[50];
    printf("Enter the string : ");
    scanf("%[^\n]c", s);
    printf("Enter the word to be searched : ");
    scanf(" %s", w);

    int slen = strlen(s);
    int wlen = strlen(w);
    char *d_s, *d_w;
    int *d_c, hcount;
    hipMalloc(&d_s, slen * sizeof(char));
    hipMalloc(&d_w, wlen * sizeof(char));
    hipMalloc(&d_c, sizeof(int));

    hipMemcpy(d_s, s, slen * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, wlen * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_c, &hcount, sizeof(int), hipMemcpyHostToDevice);

    int bs = 256, gs = (slen + bs - 1) / bs;
    countOccurrences<<<gs, bs>>>(d_s, d_w, d_c, slen, wlen);

    hipMemcpy(&hcount, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times.\n", w, hcount);

    hipFree(d_s);
    hipFree(d_w);
    hipFree(d_c);

    return 0;
}
